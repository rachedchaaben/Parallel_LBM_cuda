#include "hip/hip_runtime.h"
#include <cstdlib> // for malloc
#include <iostream>
#include <vector>
#include <sstream>

#include "LBMSolver.h"

#include "lbmFlowUtils.h"

#include "writePNG/lodepng.h"
#include "writeVTK/saveVTK.h"

#include "cuda_error.h"

#include <sys/time.h>
struct timeval t1, t2;

// ======================================================
// ======================================================
LBMSolver::LBMSolver(const LBMParams& params) :
  params(params)
{

  const int nx = params.nx;
  const int ny = params.ny;
  const int npop = LBMParams::npop;

  // memory allocations

  // distribution functions
  fin  = (real_t*) malloc(nx*ny*npop * sizeof(real_t));
  fout = (real_t*) malloc(nx*ny*npop * sizeof(real_t));
  feq  = (real_t*) malloc(nx*ny*npop * sizeof(real_t));

  hipMalloc((real_t**)&fin_d, nx*ny*npop * sizeof(real_t));
  hipMalloc((real_t**)&fout_d, nx*ny*npop * sizeof(real_t));
  hipMalloc((real_t**)&feq_d, nx*ny*npop * sizeof(real_t));

  // macroscopic variables
  rho = (real_t*) malloc(nx*ny * sizeof(real_t));
  ux  = (real_t*) malloc(nx*ny * sizeof(real_t));
  uy  = (real_t*) malloc(nx*ny * sizeof(real_t));
  
  hipMalloc((real_t**)&rho_d, nx*ny*npop * sizeof(real_t));
  hipMalloc((real_t**)&ux_d, nx*ny*npop * sizeof(real_t));
  hipMalloc((real_t**)&uy_d, nx*ny*npop * sizeof(real_t));

  // obstacle
  obstacle = (int *) malloc(nx*ny * sizeof(int));
  
  hipMalloc((real_t**)&obstacle_d, nx*ny* sizeof(int));

} // LBMSolver::LBMSolver

// ======================================================
// ======================================================
LBMSolver::~LBMSolver()
{
  // free memory

  // distribution functions
  delete[] fin;
  delete[] fout;
  delete[] feq;
  
  hipFree(fin_d);
  hipFree(fout_d);
  hipFree(feq_d);
  
  // macroscopic variables
  delete[] rho;
  delete[] ux;
  delete[] uy;
  
  hipFree(rho_d);
  hipFree(ux_d);
  hipFree(uy_d);

  // obstacle
  delete[] obstacle;
  
  hipFree(obstacle_d);
  
} // LBMSolver::~LBMSolver

// ======================================================
// ======================================================
void LBMSolver::initialize()
{

  // initialize obstacle mask array
  init_obstacle_mask(params, obstacle, obstacle_d);

  // initialize macroscopic velocity
  initialize_macroscopic_variables(params, 
                                   rho, rho_d, 
                                   ux, ux_d, 
                                   uy, uy_d);

  // Initialization of the populations at equilibrium 
  // with the given macroscopic variables.
  equilibrium(params, v, t, rho_d, ux_d, uy_d, fin_d);
  
} // LBMSolver::initialize

// ======================================================
// ======================================================
void LBMSolver::run()
{
  //Uncomment for time measurment 
  //gettimeofday(&t1, 0);

  initialize();
  
  // time loop
  for (int iTime=0; iTime<params.maxIter; ++iTime) {

    if (iTime % 100 == 0) {
      output_png(iTime);
      output_vtk(iTime);
    }

    // Right wall: outflow condition.
    // we only need here to specify distrib. function for velocities
    // that enter the domain (other that go out, are set by the streaming step)
    border_outflow(params, fin_d);
      
    // Compute macroscopic variables, density and velocity.
    macroscopic(params, v, fin_d, rho_d, ux_d, uy_d);
      
    // Left wall: inflow condition.
    border_inflow(params, fin_d, rho_d, ux_d, uy_d);

    // Compute equilibrium.
    equilibrium(params, v, t, rho_d, ux_d, uy_d, feq_d);
    update_fin_inflow(params, feq_d, fin_d);

    // Collision step.
    compute_collision(params, fin_d, feq_d, fout_d);

    // Bounce-back condition for obstacle.
    // in python language, we "slice" fout by obstacle
    update_obstacle(params, fin_d, obstacle_d, fout_d);

    // Streaming step.
    streaming(params, v, fout_d, fin_d);

  } // end for iTime
  // Uncomment for time measurment
  /*
  gettimeofday(&t2, 0);
  double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
  printf("Time to generate:  %3.1f ms \n", time);
  */
} // LBMSolver::run

// ======================================================
// ======================================================
void LBMSolver::output_png(int iTime)
{

  std::cout << "Output data (PNG) at time " << iTime << "\n";

  const int nx = params.nx;
  const int ny = params.ny;

  // copy data device to host
  hipMemcpy(ux, ux_d, nx*ny*sizeof(real_t), hipMemcpyDeviceToHost);
  hipMemcpy(uy, uy_d, nx*ny*sizeof(real_t), hipMemcpyDeviceToHost);

  real_t* u2 = (real_t *) malloc(nx*ny*sizeof(real_t));

  // compute velocity norm, as well as min and max values
  real_t min_value = sqrt(ux[0]*ux[0] + uy[0]*uy[0]);
  real_t max_value = min_value;
  for (int j = 0; j < ny; ++j) {
    for (int i = 0; i < nx; ++i) {

      int index = i + nx * j;

      u2[index] = sqrt(ux[index]*ux[index] + uy[index]*uy[index]);

      if (u2[index]<min_value)
        min_value = u2[index];

      if (u2[index]>max_value)
        max_value = u2[index];

    } // end for i

  } // end for j

  // create png image buff
  std::vector<unsigned char> image;
  image.resize(nx * ny * 4);
  for (int j = 0; j < ny; ++j) {
    for (int i = 0; i < nx; ++i) {

      int index = i + nx * j;

      // rescale velocity in 0-255 range
      unsigned char value = static_cast<unsigned char>((u2[index]-min_value)/(max_value-min_value)*255);
      image[0 + 4*i + 4*nx*j] = value; 
      image[1 + 4*i + 4*nx*j] = value; 
      image[2 + 4*i + 4*nx*j] = value; 
      image[3 + 4*i + 4*nx*j] = value; 
    }
  }

  std::ostringstream iTimeNum;
  iTimeNum.width(7);
  iTimeNum.fill('0');
  iTimeNum << iTime;

  std::string filename  = "vel_" + iTimeNum.str() + ".png";

  // encode the image
  unsigned error = lodepng::encode(filename, image, nx, ny);

  //if there's an error, display it
  if(error) std::cout << "encoder error " << error << ": "<< lodepng_error_text(error) << std::endl;


  delete[] u2;

} // LBMSolver::output_png

// ======================================================
// ======================================================
void LBMSolver::output_vtk(int iTime)
{

  std::cout << "Output data (VTK) at time " << iTime << "\n";

  bool useAscii = false; // binary data leads to smaller files
  
  const int nx = params.nx;
  const int ny = params.ny;
  
  // copy data device to host  
  hipMemcpy(ux, ux_d, nx*ny*sizeof(real_t), hipMemcpyDeviceToHost);
  hipMemcpy(ux, ux_d, nx*ny*sizeof(real_t), hipMemcpyDeviceToHost);
  hipMemcpy(rho, rho_d, nx*ny*sizeof(real_t), hipMemcpyDeviceToHost);

  saveVTK(rho, ux, uy, params, useAscii, iTime);

} // LBMSolver::output_vtk
