#include "hip/hip_runtime.h"
#include <math.h> // for M_PI = 3.1415....

#include "lbmFlowUtils.h"

#include "lbmFlowUtils_kernels.h"
#include "cuda_error.h"

const int nbThreads = 32;

// ======================================================
// ======================================================
void macroscopic(const LBMParams& params, 
                 const velocity_array_t v,
                 const real_t* fin_d,
                 real_t* rho_d,
                 real_t* ux_d,
                 real_t* uy_d)
{

  const int nx = params.nx;
  const int ny = params.ny;

  // call kernel
  dim3 grid_size((nx+nbThreads-1)/nbThreads,(ny+nbThreads-1)/nbThreads);
  dim3 block_dim(nbThreads, nbThreads);
  
  macroscopic_kernel<<<grid_size,block_dim>>>(params,v,fin_d,rho_d,ux_d,uy_d);

} // macroscopic

// ======================================================
// ======================================================
void equilibrium(const LBMParams& params, 
                 const velocity_array_t v,
                 const weights_t t,
                 const real_t* rho_d,
                 const real_t* ux_d,
                 const real_t* uy_d,
                 real_t* feq_d)
{

  const int nx = params.nx;
  const int ny = params.ny;

  // call kernel 
  dim3 grid_size((nx+nbThreads-1)/nbThreads,(ny+nbThreads-1)/nbThreads);
  dim3 block_dim(nbThreads, nbThreads);
  
  equilibrium_kernel<<<grid_size,block_dim>>>(params,v,t,rho_d,ux_d,uy_d,feq_d);
  
} // equilibrium

// ======================================================
// ======================================================
void init_obstacle_mask(const LBMParams& params, 
                        int* obstacle, 
                        int* obstacle_d)
{

  const int nx = params.nx;
  const int ny = params.ny;

  const real_t cx = params.cx;
  const real_t cy = params.cy;

  const real_t r = params.r;

  for (int j = 0; j < ny; ++j) {
    for (int i = 0; i < nx; ++i) {

      int index = i + nx * j;

      real_t x = 1.0*i;
      real_t y = 1.0*j;

      obstacle[index] = (x-cx)*(x-cx) + (y-cy)*(y-cy) < r*r ? 1 : 0;

    } // end for i
  } // end for j

  // copy host to device
  hipMemcpy(obstacle_d,obstacle,nx*ny*sizeof(int),hipMemcpyHostToDevice);

} // init_obstacle_mask

// ======================================================
// ======================================================
__host__ __device__
real_t compute_vel(int dir, int i, int j, real_t uLB, real_t ly)
{

  // flow is along X axis
  // X component is non-zero
  // Y component is always zero

  return (1-dir) * uLB * (1 + 1e-4 * sin(j/ly*2*M_PI));

} // compute_vel

// ======================================================
// ======================================================
void initialize_macroscopic_variables(const LBMParams& params, 
                                      real_t* rho, real_t* rho_d,
                                      real_t* ux, real_t* ux_d,
                                      real_t* uy, real_t* uy_d)
{

  const int nx = params.nx;
  const int ny = params.ny;

  for (int j = 0; j < ny; ++j) {
    for (int i = 0; i < nx; ++i) {

      int index = i + nx * j;

      rho[index] = 1.0;
      ux[index]  = compute_vel(0, i, j, params.uLB, params.ly);
      uy[index]  = compute_vel(1, i, j, params.uLB, params.ly);

    } // end for i
  } // end for j

  // copy host to device
  hipMemcpy(rho_d,rho,nx*ny*sizeof(real_t),hipMemcpyHostToDevice);
  hipMemcpy(ux_d,ux,nx*ny*sizeof(real_t),hipMemcpyHostToDevice);
  hipMemcpy(uy_d,uy,nx*ny*sizeof(real_t),hipMemcpyHostToDevice);

} // initialize_macroscopic_variables

// ======================================================
// ======================================================
void border_outflow(const LBMParams& params, real_t* fin_d)
{
  const int ny = params.ny;
  
  // call kernel
  dim3 grid_size((ny+nbThreads-1)/nbThreads);
  dim3 block_dim(nbThreads);
  border_outflow_kernel<<<grid_size,block_dim>>>(params,fin_d);
  

} // border_outflow

// ======================================================
// ======================================================
void border_inflow(const LBMParams& params, const real_t* fin_d, 
                   real_t* rho_d, real_t* ux_d, real_t* uy_d)
{
  const int ny = params.ny;
  
  // call kernel
  dim3 grid_size((ny+nbThreads-1)/nbThreads);
  dim3 block_dim(nbThreads);
  
  border_inflow_kernel<<<grid_size,block_dim>>>(params,fin_d,rho_d,ux_d,uy_d);
  
} // border_inflow

// ======================================================
// ======================================================
void update_fin_inflow(const LBMParams& params, const real_t* feq_d, 
                       real_t* fin_d)
{

  const int ny = params.ny;
  
  // call kernel
  dim3 grid_size((ny+nbThreads-1)/nbThreads);
  dim3 block_dim(nbThreads);
  
  update_fin_inflow_kernel<<<grid_size,block_dim>>>(params,feq_d,fin_d);
  
} // update_fin_inflow
  
// ======================================================
// ======================================================
void compute_collision(const LBMParams& params, 
                       const real_t* fin_d,
                       const real_t* feq_d,
                       real_t* fout_d)
{

  const int nx = params.nx;
  const int ny = params.ny;
  
  // call kernel
  dim3 grid_size((nx+nbThreads-1)/nbThreads,(ny+nbThreads-1)/nbThreads);
  dim3 block_dim(nbThreads,nbThreads);
  
  compute_collision_kernel<<<grid_size,block_dim>>>(params,fin_d,feq_d,fout_d);
  
} // compute_collision

// ======================================================
// ======================================================
void update_obstacle(const LBMParams &params, 
                     const real_t* fin_d,
                     const int* obstacle_d, 
                     real_t* fout_d)
{

  const int nx = params.nx;
  const int ny = params.ny;

  // call kernel
  dim3 grid_size((nx+nbThreads-1)/nbThreads,(ny+nbThreads-1)/nbThreads);
  dim3 block_dim(nbThreads,nbThreads);
  
  update_obstacle_kernel<<<grid_size,block_dim>>>(params,fin_d,obstacle_d,fout_d);
} // update_obstacle

// ======================================================
// ======================================================
void streaming(const LBMParams& params,
               const velocity_array_t v,
               const real_t* fout_d,
               real_t* fin_d)
{

  const int nx = params.nx;
  const int ny = params.ny;

  // call kernel  
  dim3 grid_size((nx+nbThreads-1)/nbThreads,(ny+nbThreads-1)/nbThreads);
  dim3 block_dim(nbThreads,nbThreads);
  
  streaming_kernel<<<grid_size,block_dim>>>(params,v,fout_d,fin_d);


} // streaming
